#include "hip/hip_runtime.h"
/*
 * SortingAndSearch_cuda.cu
 *
 *  Created on: 19/03/2016
 *      Author: bruno
 */

#include "SortingAndSearch.h"
#include "aux.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "World.cuh"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>

#define EMPTY 0xffffffff

extern __constant__
SysParams system_params;

struct cell_before
{
	__host__ __device__
	bool operator()(const uint4 &left, const uint4 &right) const
	{
		return (left.z < right.z) || (left.z == right.z && left.y < right.y) ||
				(left.z == right.z && left.y == right.y && left.x < right.x);
	}
};

struct cell_equal
{
	__host__ __device__
	bool operator()(const uint4 &left, const uint4 &right) const
	{
		return left.z == right.z && left.y == right.y && left.x == right.x;
	}
};


__host__ __device__
uint find_first(uint4 *dSortedGrid, const uint4 value, const uint n){
	uint l = 0;
	uint r = n - 1;

	while( l <= r){
		unint m = (l + r)/2;
		if( cell_equal(dSortedGrid[m], value)){ // found, search for first
			while(m > 0 && cell_equal(dSortedGrid[m-1], dSortedGrid[m]))
				m--;
			return m;

		}else if(cell_before(dSortedGrid[m]), value){
			l = m + 1;
		}else{
			r = m - 1;
		}
	}

	return n;
}


__global__
void prepare_grid(uint4 *dSortedGrid, float4 *dPos,
		unsigned int n_particles, float3 p_min, float d)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx>=n_particles) return;

	int3 gridPos = get_grid_pos(make_float3(dPos[idx]), p_min, d);

	dSortedGrid[idx].x = gridPos.x;
	dSortedGrid[idx].y = gridPos.y;
	dSortedGrid[idx].z = gridPos.z;
	dSortedGrid[idx].w = idx;

}


void SortingAndSearch::prepareGrid(float4 *dPos){
	unsigned int numThreads, numBlocks;
	computeGridSize(n_particles, 256, &numBlocks, &numThreads);

	prepare_grid<<<numBlocks, numThreads>>>(dSortedGrid, dPos, n_particles,
			p_min, d);

	getLastCudaError("Kernel execution failed: prepare_grid");
}

void SortingAndSearch::sortParticles(){
	thrust::sort(thrust::device_ptr<uint4>(dSortedGrid),
			thrust::device_ptr<uint4>(dSortedGrid + n_particles),
			cell_before());
}

__global__
void reorder_pos_vel(float4 *sortedPos, float4 *sortedVel,
		uint4 *dSortedGrid, float4 *oldPos, float4 *oldVel,
        unsigned int n_particles)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= n_particles) return;

	uint p_id = dSortedGrid[idx].w;

	sortedPos[idx] = oldPos[p_id];
	sortedVel[idx] = oldVel[p_id];
}

void SortingAndSearch::reorderPosAndVel(float4 *dPos, float4 *dVel){
	unsigned int numBlocks, numThreads;
	computeGridSize(n_particles, 256, &numBlocks, &numThreads);

	reorder_pos_vel<<<numBlocks, numThreads>>>(dSortedPos, dSortedVel,
			dSortedGrid, dPos, dVel, n_particles);

	getLastCudaError("Kernel execution failed: reorder_pos_vel");
}

__global__
void calculate_contact_force(float4 *sortedPos, float4 *sortedVel,
		uint4 *dSortedGrid, float4 *force, unsigned int n_particles,
		float3 pMin, float d, int3 gridDim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= n_particles) return;

	float3 pos = make_float3(sortedPos[idx]);
	float3 vel = make_float3(sortedVel[idx]);

	int3 gridPos = get_grid_pos(pos, pMin, d);

	float3 resulting_force = make_float3(0);

	float r = system_params.particle_radius;

	for(int z = -1; z <= 1; z++){
		for(int y = -1; y <= 1; y++){
			// get index of first x-1 continue until x != x+1
			
			int x = gridPos.x > 0 ? gridPos.x-1: gridPos.x;
			uint4 cell = make_uint4(x, gridPos.y + y, gridPos.z + z,0);
			
			if(cell.y >= gridDim.y || cell.z >= gridDim.z)
				continue;

			uint other = find_first(dSortedGrid,cell, n_particles);
			while(other < n_particles){
				uint4 neigh = dSortedGrid[other];
				if(neigh.z != cell.z || neigh.y != cell.y || neigh.x > cell.x + 1){
					// Check if next one is still in line (x-1 -> x+1, y, z)
					break;
				}
				if(neigh.w != idx){
					float3 neigh_pos = make_float3(sortedPos[other]);
					float3 neigh_vel = make_float3(sortedVel[other]);

					resulting_force += World::contactForce(pos, neigh_pos,
							vel, neigh_vel, r, r);
				}
				other++;
			}
		}
	}

	force[dSortedGrid[idx].w] = make_float4(resulting_force);
}

void SortingAndSearch::calculateContactForce(float4 *dPos, float4 *dVel, float4 *dFor){
	// will not use dPos and dVel since i have my own version stored
	unsigned int numBlocks, numThreads;
	computeGridSize(n_particles, 256, &numBlocks, &numThreads);

	calculate_contact_force<<<numBlocks, numThreads>>>(dSortedPos, dSortedVel,
			dSortedGrid, dFor, n_particles, p_min, d, gridSize);

	getLastCudaError("Kernel execution failed: calculate_contact_force");
}



