#include "hip/hip_runtime.h"
/*
 * DirectMapping_cuda.cu
 *
 *  Created on: 09/02/2016
 *      Author: bruno
 */

#include "DirectMapping.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "aux.h"
#include "World.cuh"
#include <hip/hip_runtime.h>

extern __constant__
SysParams system_params;

__global__
void create_neighboor_grid(float4 *pos, int *grid_list, int *grid_count,
		unsigned int n_particles, float3 p_min, float d, int3 gridDim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= n_particles)
		return;

	int3 gridPos = get_grid_pos(make_float3(pos[idx]), p_min, d);

	int grid_idx = pos_to_index(gridPos, gridDim);

	int list_idx = atomicAdd(&grid_count[grid_idx], 1);
	grid_list[grid_idx*CELL_MAX_P + list_idx] = idx;

}

void DirectMapping::createNeighboorList(float4 *dPos, float4 *dVel){
	checkCudaErrors(hipMemset(dGridCounter, 0, sizeof(int) * gridDim.x * gridDim.y * gridDim.z));

	unsigned int numBlocks, numThreads;
	computeGridSize(n_particles, 256, &numBlocks, &numThreads);

	create_neighboor_grid<<<numBlocks, numThreads>>>(dPos, dGrid, dGridCounter,
			n_particles, p_min, d, gridDim);

	getLastCudaError("Kernel execution failed: create_neighboor_grid");
}

__global__
void dm_calculate_contact_force(int *grid_list, int *grid_count, float4 *pos,
		float4 *vel, float4 *force, unsigned int n_particles, float3 pMin,
		float d, int3 gridDim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= n_particles)
		return;


	float3 resulting_force = make_float3(0);
    float3 my_pos = make_float3(pos[idx]);
    float3 my_vel = make_float3(vel[idx]);

    int3 gridPos = get_grid_pos(my_pos, pMin, d);
	int grid_idx = pos_to_index(gridPos, gridDim);
	int r = system_params.particle_radius;

	for(int z = -1; z <= 1; z++){
		for(int y = -1; y <= 1; y++){
			for(int x = -1; x <= 1; x++){
				// check boundaries
				int3 other_cell_pos = gridPos + make_int3(x,y,z);
				if(other_cell_pos.x < 0 || other_cell_pos.y < 0 ||
						other_cell_pos.z < 0 ||other_cell_pos.x >= gridDim.x ||
						other_cell_pos.y >= gridDim.y ||
						other_cell_pos.x >= gridDim.z)
				{
					continue;
				}

				int other_cell = pos_to_index(other_cell_pos, gridDim);
				for(int i = 0; i < grid_count[other_cell]; i++){
					int p_index = grid_list[other_cell*CELL_MAX_P + i - 1];
					if(p_index != idx)
						resulting_force += World::contactForce(my_pos,
													make_float3(pos[p_index]),
													my_vel,
													make_float3(vel[p_index]),
													r, r);
				}
			}
		}
	}
	force[idx] = make_float4(resulting_force);

}

void DirectMapping::calculateContactForce(float4 *dPos, float4 *dVel, float4 *dFor){
	unsigned int numBlocks, numThreads;
	computeGridSize(n_particles, 256, &numBlocks, &numThreads);

	dm_calculate_contact_force<<<numBlocks, numThreads>>>(dGrid, dGridCounter,
			dPos, dVel, dFor, n_particles, p_min, d, gridDim);

	getLastCudaError("Kernel execution failed: dm_calculate_contact_force");
}

