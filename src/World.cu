#include "hip/hip_runtime.h"
/*
 * World.cpp
 *
 *  Created on: 29/01/2016
 *      Author: bruno
 */

#include "World.cuh"
#include "ParticleSystem.cuh"
#include "hip/hip_vector_types.h"

__device__
void World::checkBoudaries(float4* pos, float4* vel)
{
	// inicialmente vamos apenas impedir as particulas de passarem
	// pelo chão
	if(pos->z < -1.0){
		pos->z = -1.0 + system_params.particle_radius;
		vel->z *= system_params.boundarie_damping;
	}
}

__device__
float3 World::contactForce(float3 posA, float3 posB,
		float3 velA, float3 velB,
        float radiusA, float radiusB)
{
	float3 relPos = posB - posA;

	float dist = length(relPos);
	float collideDist = radiusA + radiusB;

	float3 force = make_float3(0);

	if(dist < collideDist){
		float3 norm = relPos / dist;

		// relative velocity
		float3 relVel = velB - velA;

		// relative tangential velocity
		float3 tanVel = relVel - (dot(relVel, norm) * norm);

		// spring force // spring = 0.5
		force = -0.5*(collideDist - dist) * norm;
		// dashpot (damping) force
		force += system_params.global_damping*relVel;
		// tangential shear force //shear = 0.1
		force += 0.1*tanVel;

	}

	return force;
}

