#include "hip/hip_runtime.h"

#include <iostream>

#include "ParticleSystem.h"
#include "ParticleSystem.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "World.cuh"
#include "aux.h"

__constant__
SysParams system_params;

__global__
void integrate_system(float4 *pos, float4 *vel, float4 *force, float4 *obstacles,
		unsigned int n_particles, unsigned int n_obstacles)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx>=n_particles) return; 

	float3 obs_force = make_float3(0);
	for(int i = 0; i < n_obstacles; i++){
		obs_force += World::contactForce(make_float3(pos[idx]), make_float3(obstacles[i]),
				make_float3(vel[idx]), make_float3(0), system_params.particle_radius,
				obstacles[i].z);
	}
	force[idx] += make_float4(obs_force);

	float3 vel_f = make_float3(vel[idx] + force[idx]);
	vel_f += system_params.gravity*system_params.dt;
	vel_f *= system_params.global_damping;

	pos[idx] += make_float4(vel_f*system_params.dt) ;
	vel[idx] =  make_float4(vel_f);

	World::checkBoudaries(&pos[idx], &vel[idx]);

}

void ParticleSystem::integrate(){
	unsigned int n_threads, n_blocks;
	computeGridSize(params.n_particles,256, &n_blocks, &n_threads);
	integrate_system<<< n_blocks, n_threads >>>(dPos, dVel, dFor, dObs, params.n_particles, params.n_obstacles);
}

void ParticleSystem::copyParticlesToDevice(){
        checkCudaErrors(hipMemcpy(dPos, hPos, sizeof(float4)*params.n_particles,
        							hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dVel, hVel, sizeof(float4)*params.n_particles,
        							hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dObs, hObs, sizeof(float4)*params.n_obstacles,
                							hipMemcpyHostToDevice));

}

void ParticleSystem::copyParticlesToHost(){
        checkCudaErrors(hipMemcpy(hPos, dPos, sizeof(float4)*params.n_particles,
        							hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(hVel, dVel, sizeof(float4)*params.n_particles,
        							hipMemcpyDeviceToHost));

}

