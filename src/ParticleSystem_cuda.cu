#include "hip/hip_runtime.h"

#include <iostream>

#include "ParticleSystem.h"
#include "ParticleSystem.cuh"
#include "hip/hip_runtime_api.h"

__global__
void integrate_system(float4 *pos, float4 *vel,
						float dt, unsigned int n_particles,
						float damping)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx>=n_particles) return; 
	
	pos[idx].x = idx;
	pos[idx].y = idx*idx;

}

void ParticleSystem::integrate(){
	unsigned int n_threads, n_blocks;
	computeGridSize(n_particles,256, &n_blocks, &n_threads);
	integrate_system<<< n_blocks, n_threads >>>(dPos, dVel, dt, n_particles, 1); //TODO: Set damping
}

void ParticleSystem::copyParticlesToDevice(){
        checkCudaErrors(hipMemcpy(dPos, hPos, sizeof(float4)*n_particles, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dVel, hVel, sizeof(float4)*n_particles, hipMemcpyHostToDevice));

}

void ParticleSystem::copyParticlesToHost(){
        checkCudaErrors(hipMemcpy(hPos, dPos, sizeof(float4)*n_particles, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(hVel, dVel, sizeof(float4)*n_particles, hipMemcpyDeviceToHost));

}
