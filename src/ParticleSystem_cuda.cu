#include "hip/hip_runtime.h"

#include <iostream>

#include "ParticleSystem.h"
#include "ParticleSystem.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "World.cuh"

__constant__
SysParams system_params;

__global__
void integrate_system(float4 *pos, float4 *vel, unsigned int n_particles)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx>=n_particles) return; 
	//TODO Calcular antes força resultante para cada partícula (incluindo
	// aceleração gravitacional)
	pos[idx] += vel[idx]*system_params.dt*system_params.global_damping;

	World::checkBoudaries(&pos[idx], &vel[idx]);

}

void ParticleSystem::integrate(){
	unsigned int n_threads, n_blocks;
	computeGridSize(n_particles,256, &n_blocks, &n_threads);
	integrate_system<<< n_blocks, n_threads >>>(dPos, dVel, n_particles);
}

void ParticleSystem::copyParticlesToDevice(){
        checkCudaErrors(hipMemcpy(dPos, hPos, sizeof(float4)*n_particles,
        							hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dVel, hVel, sizeof(float4)*n_particles,
        							hipMemcpyHostToDevice));

}

void ParticleSystem::copyParticlesToHost(){
        checkCudaErrors(hipMemcpy(hPos, dPos, sizeof(float4)*n_particles,
        							hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(hVel, dVel, sizeof(float4)*n_particles,
        							hipMemcpyDeviceToHost));

}
