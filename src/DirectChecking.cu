#include "hip/hip_runtime.h"
/*
 * DirectChecking_cuda.cu
 *
 *  Created on: 02/02/2016
 *      Author: bruno
 */

#include <hip/hip_runtime.h>

#include "DirectChecking.h"
#include "ParticleSystem.h"
#include "World.cuh"
#include "aux.h"

extern __constant__
SysParams system_params;

__global__
void calculate_contact_force(thrust::host_vector<float4>& dPos, thrust::host_vector<float4>& dVel, 
                                thrust::host_vector<float4>& dFor){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int n_particles = dPos.size();
        if(idx>=n_particles) return;

        float3 force = make_float3(0,0,0);
        float r = system_params.particle_radius;
        float3 position = make_float3(dPos[idx]);
        float3 velocity = make_float3(dVel[idx]);
        for(int i = 0; i < n_particles; i++){
                if(i != idx)
                    force += World::contactForce(position, make_float3(dPos[i]), velocity, make_float3(dVel[i]), r, r);
        }
        dFor[idx] = make_float4(force, 0);
}

void DirectChecking::calculateContactForce(thrust::host_vector<float4>& dPos, thrust::host_vector<float4>& dVel, 
                                                thrust::host_vector<float4>& dFor){
	unsigned int n_threads, n_blocks;
        unsigned int n_particles = dPos.size();
	computeGridSize(n_particles,256, &n_blocks, &n_threads);
	calculate_contact_force<<<n_blocks, n_threads>>>(dPos, dVel, dFor);
}


